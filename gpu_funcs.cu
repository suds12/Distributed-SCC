#include <iostream>

#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <stdgpu/memory.h>          // createDeviceArray, destroyDeviceArray
#include <stdgpu/iterator.h>        // device_begin, device_end
#include <stdgpu/platform.h>        // STDGPU_HOST_DEVICE
#include <stdgpu/unordered_map.cuh> // stdgpu::unordered_map
#include <stdgpu/vector.cuh>        // stdgpu::vector

#include "kernals.h"



void cuda_h2d(int* src, int* dst, int size)
{

	hipMallocManaged(&dst, size*sizeof(int));
	hipMemcpy(dst, src, size*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(temp, dst, size*sizeof(int), hipMemcpyDeviceToHost);
}

void cuda_d2h(int* src, int* dst, int size)
{
	//hipMallocManaged(&src, size*sizeof(int));
	hipMemcpy(src, dst, size*sizeof(int), hipMemcpyDeviceToHost);
	
	
}

void deallocate_device_mem(int* ptr)
{
	hipFree(ptr);

} 

void cuda_sync()
{
	hipDeviceSynchronize();
}





