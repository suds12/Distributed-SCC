#include <iostream>

#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <stdgpu/memory.h>          // createDeviceArray, destroyDeviceArray
#include <stdgpu/iterator.h>        // device_begin, device_end
#include <stdgpu/platform.h>        // STDGPU_HOST_DEVICE
#include <stdgpu/unordered_map.cuh> // stdgpu::unordered_map
#include <stdgpu/vector.cuh>        // stdgpu::vector

#include "kernals.h"



void cuda_h2d(int* src, int* dst, int size)
{

	hipMalloc(&dst, size*sizeof(int));
	hipMemcpy(dst, src, size*sizeof(int), hipMemcpyHostToDevice);



}

void deallocate_device_mem(int* ptr)
{
	hipFree(ptr);

}

void cuda_sync()
{
	hipDeviceSynchronize();
}

void create_MN_vector(int* meta_nodes, int size)
{
	hipMalloc(&meta_nodes, size*sizeof(int));

	
}